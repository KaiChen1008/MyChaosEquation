#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <sstream>
#include <cassert>
#include <fstream>
#include <cfloat>
#include <cstdlib>
#include <iostream>
#include <string>
#include <ctime>
#include <pthread.h>

// #include <png.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

//Global constants
static const int num_params = 18;
static const int iters = 800;
static const int steps_per_frame = 500;
static const double delta_per_step = 1e-5;
static const double delta_minimum = 1e-7;
static const double t_start = -3.0;
static const double t_end = 3.0;
static const int fad_speed = 10;
static std::mt19937 rand_gen;
static const float dot_sizes[3] = { 1.0f, 3.0f, 10.0f };

//Global variables
static int window_w = 1600;
static int window_h = 900;
static int window_bits = 24;
static float plot_scale = 0.25f;
static float plot_x = 0.0f;
static float plot_y = 0.0f;


double params[num_params];              // 18 
int start_point[6] = {0, 1, 2, 3, 4, 5};


pthread_mutex_t t_mutex   = PTHREAD_MUTEX_INITIALIZER;
int num_threads = 6;

struct Color{
    int r;
    int g;
    int b;
};

struct Vector2f{
    double x;
    double y;
} ;

struct Vertex{
    Vector2f position;
    Color  color;
};



static Color GetRandColor(int i) {
  i += 1;
  int r = std::min(255, 50 + (i * 11909) % 256);
  int g = std::min(255, 50 + (i * 52973) % 256);
  int b = std::min(255, 50 + (i * 44111) % 256);
  return Color{r, g, b};
}

static void ResetPlot() {
  plot_scale = 0.25f;
  plot_x = 0.0f;
  plot_y = 0.0f;
}

static Vector2f ToScreen(double x, double y) {
  const float s = plot_scale * float(window_h / 2);
  const float nx = float(window_w) * 0.5f + (float(x) - plot_x) * s;
  const float ny = float(window_h) * 0.5f + (float(y) - plot_y) * s;
  return Vector2f{nx, ny};
}

static void RandParams(double* params) {
	params[ 0] = 1; params[ 1] = 0; params[ 2] = 0;
	params[ 3] = 0; params[ 4] =-1; params[ 5] = 1;
	params[ 6] =-1; params[ 7] = 0; params[ 8] = 0;

	params[ 9] = 0; params[10] =-1; params[11] =-1;
	params[12] =-1; params[13] =-1; params[14] =-1;
	params[15] = 0; params[16] =-1; params[17] = 0;
}

static void draw_png() {
    // write_png(output);
	// dot_size
	// window.draw(vertex_array.data(), vertex_array.size(), sf::PrimitiveType::Points);
}

void* thread_target(void* arg) {
    int* start = (int*) arg;
    double t =  double(start[0]);
    
    // Setup the vertex array
    vector<Vertex> vertex_array(iters * steps_per_frame); // 800 * 500
    for (size_t i = 0; i < vertex_array.size(); ++i) 
        vertex_array[i].color = GetRandColor(i % iters);


    for (; t < t+1.0; )
    {
        for (int step = 0; step < steps_per_frame; ++step) //steps = 500
        {
			bool isOffScreen = true;
            double x = t;
            double y = t;
            for (int iter = 0; iter < iters; ++iter) // 800
            {
                const double xx = x * x; const double yy = y * y; const double tt = t * t;
                const double xy = x * y; const double xt = x * t; const double yt = y * t;

                const double nx = xx * params[ 0] + yy * params[ 1] + tt * params[ 2] + 
                            xy * params[ 3] + xt * params[ 4] + yt * params[ 5] + 
                            x  * params[ 6] + y  * params[ 7] + t  * params[ 8] ;
                
                const double ny = xx * params[ 9] + yy * params[10] + tt * params[11] + 
                            xy * params[12] + xt * params[13] + yt * params[14] + 
                            x  * params[15] + y  * params[16] + t  * params[17] ;

                x = nx;
                y = ny;

				Vector2f screenPt = ToScreen(x,y);
                if (iter < 100)
                {
                    screenPt.x = FLT_MAX;
                    screenPt.y = FLT_MAX;
                }

                vertex_array[step*iters + iter].position = screenPt;


            } //iteration end

			t += 1e-7;
        } // step end


		// Draw the data
		draw_png();

    } // t end

}

int main(int argc, char* argv[]) {
	clock_t start, stop;
	cout << "start computing........." << endl;
	start = clock();


    // Initialize random parameters
    ResetPlot();
    RandParams(params);

    
    pthread_t threads[num_threads];

    for (int i = 0; i < num_threads; ++i)
    {
        assert (0 == pthread_create(&threads[i], NULL, thread_target, (void*) &start_point[i]);
    }
        

    for (int i = 0; i< num_threads; ++i)
    {
        assert(0 == pthread_join(threads[i], NULL));
    }

	stop = clock();
	cout << double(stop - start) / CLOCKS_PER_SEC << endl;
    return 0;
}



